#include "hip/hip_runtime.h"
/*
 * File:     it_mult_vec_test.cu
 *
 * Purpose:  Test matrix vector multiplication y=Ax.
 *           Matrix A is a square matrix of size nxn.
 *           Column vectors x and y are of size nx1
 *
 * Input:    A[i][j]=c in all positions.  y[i] is 0 in all positions
 *           x[i]= i for 0<=i<n
 *
 * Note:     For simplicity, we assume n is divisible by no_proc
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "minunit.h"
#include "it_mult_vec.h"

#define MAX_TEST_MATRIX_SIZE 2048

#define FAIL 0
#define SUCC 1

#define TEST_CORRECTNESS 1

const double ERROR = 0.0001;

void print_error(const char *msgheader, const char *msg){
  printf("%s error: %s\n", msgheader, msg);
}

/*-------------------------------------------------------------------
 * Initialize test matrix and vectors.
 *   vector x of size n:    0 for every element
 *   vector d of size n:    (2n-1)/n for every element
 *
 * When matrix_type is not UPPER_TRIANGULAR
 *   matrix A of size nxn:  A[i,i]=0 for diagnal elements;
 *                          A[i,j]=-1/n for non-diagonal elements
 * When matrix_type is UPPER_TRIANGULAR
 *   matrix A of size nxn:  A[i,i]=0 for diagnal elements;
 *                          A[i,j]=-1/n for upper diagonal elements with i<j,
 *                          all lower triangular elements are 0
 * In args:
 *   n is the number of columns (and rows)
 *   marix_type: matrix type UPPER_TRIANGULAR or not
 *
 * Return value:
 *   If failed, return FAIL (0)
 *   If successful, return SUCC (1)
 */
int init_matrix(
    float *x, float *y, float *d, float *A, float *diff, int n, int matrix_type)
{
  int i, j;

  if (x == NULL || y == NULL || d==NULL
      ||  A == NULL || diff == NULL || n<=0) return FAIL;

  for (i = 0; i < n; i++) {
    x[i]=0;
    if (matrix_type == UPPER_TRIANGULAR) {
      d[i] = (2.0 * n - 1.0 * i - 1.0) / n;
    } else {
      d[i] = (2.0 * n - 1.0) / n;
    }

    for (j = 0; j < i; j++)
      if (matrix_type == UPPER_TRIANGULAR) {
        A[i * n + j] = 0.0;
      } else {
        A[i * n + j] = -1.0 / n;
      }

    A[i * n + i] = 0.0;

    for (j = i + 1; j < n; j++) {
        A[i * n + j] = -1.0 / n;
    }
  }

  return SUCC;
}

/*-----------------------------------
 * Validate the correctness of iterative computation
 * For the non-asynchrous mode, we use the Jacobi method, and thus you can use any thread configuration
 * For the asynchrous mode, we use the Gauss-Seidel method, you have to use 1x1 thread  configuration.
 *           Namely one block with 1 thread. With asychnous Gauss-Seidel running on multiple threads,
 *           the update speed may be inconsistent from one run to another run.
 */
 
const char*  validate_vect(
  const char *msgheader, float *actual_y, int n, int t, int matrix_type, int use_async)
{
  int i;
  if(n <= 0 )
    return "Failed: 0 or negative size";
  if(n > MAX_TEST_MATRIX_SIZE)
    return "Failed: Too big to validate";

  // Calculate expected.
  float *A, *x, *d, *y, *diff;

  A = (float*)malloc(n*n*sizeof(float));
  x = (float*)malloc(n*sizeof(float));
  d = (float*)malloc(n*sizeof(float));
  y = (float*)malloc(n*sizeof(float));
  diff = (float*)malloc(n*sizeof(float));
  init_matrix( x, y, d, A, diff, n, matrix_type);
  if(use_async)
    gsit_mult_vec_seq(n, y, d, A, x, matrix_type, t);
  else
    it_mult_vec_seq(n, y, d, A, x, matrix_type, t);

  for (i = 0; i < n; i++){
#ifdef DEBUG1
    printf("%s i=%d  Expected %f Actual %f\n", msgheader, i, y[i], actual_y[i]);
#endif
    mu_assert(
      "One mismatch in iterative mat-vect multiplication",
      fabs(y[i] - actual_y[i]) <= ERROR );
  }

  free(A);
  free(x);
  free(y);
  free(d);
  free(diff);
  return NULL;
}

/*-------------------------------------------------------------------
 * Test matrix vector multiplication
 * Process 0 collects the  error detection. If failed, return a message string
 * If successful, return NULL
 */
const char * itmv_test(
  const char *testmsg, int test_correctness, int n, int matrix_type, int t,
  int num_blocks, int threads_per_block, int use_async, int use_shared_x)
{
  float *A, *x, *d, *y, *diff;
  const char *msg;
  int i;

  A = (float*)malloc(n*n*sizeof(float));
  x = (float*)malloc(n*sizeof(float));
  d = (float*)malloc(n*sizeof(float));
  y = (float*)malloc(n*sizeof(float));
  diff = (float*)malloc(n*sizeof(float));
  init_matrix( x, y, d, A, diff, n, matrix_type);

  double tBefore = get_time();
  int no_iter= it_mult_vec(n, num_blocks, threads_per_block,
      y, d, A, x, diff, t, use_async, use_shared_x);
  double tAfter = get_time();

  printf("\n%s:", testmsg);
  printf("\nWith totally %d*%d threads, matrix size being %d, t being %d\n",
         num_blocks, threads_per_block, n, t);
  printf("Time cost in seconds: %f\n", tAfter - tBefore);

  float max_error = 0;
  for (i = 0; i < n; i++) 
    if (max_error < diff[i]) 
      max_error = diff[i];
  printf("Final error (|y-x|): %f.\n", max_error);
  printf("# of iterations executed: %d.\n", no_iter);
  if(no_iter<t) { 
    printf("Early exit due to convergence, even asked for %d iterations.\n", t);
    if(use_async)
      printf("Asynchronous code actually runs %d iterations.\n", no_iter);
  }
  printf("Final y[0]=%f. y[n-1]=%f\n", y[0], y[n-1]);

  msg = NULL;
  if (test_correctness == TEST_CORRECTNESS){
    msg = validate_vect(testmsg, y, n, no_iter, matrix_type, use_async);

    if (msg != NULL) print_error(testmsg, msg);
  }

  free(A);
  free(x);
  free(y);
  free(d);
  free(diff);

  return msg;
}

const char * itmv_test1() {
  return itmv_test(
      "Test 1:n=4, t=1, 1x2 threads", TEST_CORRECTNESS, 4, !UPPER_TRIANGULAR, 1, 1, 2,
      !USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test2() {
  return itmv_test(
      "Test 2:n=4, t=2, 1x2 threads", TEST_CORRECTNESS, 4, !UPPER_TRIANGULAR, 2, 1, 2,
      !USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test3() {
  return itmv_test(
      "Test 3:n=8, t=1, 1x2 threads", TEST_CORRECTNESS, 8, !UPPER_TRIANGULAR, 1, 1, 2,
      !USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test4() {
  return itmv_test(
      "Test 4:n=8, t=2, 1x2 threads", TEST_CORRECTNESS, 8, !UPPER_TRIANGULAR, 2, 1, 2,
      !USE_ASYNC, !USE_SHARED_X);
}


const char * itmv_test8a() {
  return itmv_test(
      "Test 8a:n=4, t=1, 1x1 threads/Gauss-Seidel", TEST_CORRECTNESS, 4, !UPPER_TRIANGULAR, 1, 1, 1,
      USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test8b() {
  return itmv_test(
      "Test 8b:n=4, t=2, 1x1 threads/Gauss-Seidel", TEST_CORRECTNESS, 4, !UPPER_TRIANGULAR, 2, 1, 1,
      USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test8c() {
  return itmv_test(
      "Test 8c:n=8, t=1, 1x1 threads/Gauss-Seidel", TEST_CORRECTNESS, 8, !UPPER_TRIANGULAR, 1, 1, 1,
      USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test8d() {
  return itmv_test(
      "Test 8d:n=8, t=2, 1x1 threads/Gauss-Seidel", TEST_CORRECTNESS, 8, !UPPER_TRIANGULAR, 2, 1, 1,
      USE_ASYNC, !USE_SHARED_X);
}

const char * itmv_test9() {
  return itmv_test(
      "Test 9: n=4K t=1K 32x128 threads", !TEST_CORRECTNESS,
      4096, !UPPER_TRIANGULAR, 1024, 1<<5, 1<<7, !USE_ASYNC, !USE_SHARED_X);
}

const char * itmv_test9a() {
  return itmv_test(
      "Test 9a: n=4K t=1K 16x128 threads", !TEST_CORRECTNESS,
      4096, !UPPER_TRIANGULAR, 1024, 1<<4, 1<<7, !USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test9b() {
  return itmv_test(
      "Test 9b: n=4K t=1K 8x128 threads", !TEST_CORRECTNESS,
      4096, !UPPER_TRIANGULAR, 1024, 1<<3, 1<<7, !USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test9c() {
  return itmv_test(
      "Test 9c: n=4K t=1K 4x128 threads", !TEST_CORRECTNESS,
      4096, !UPPER_TRIANGULAR, 1024, 1<<2, 1<<7, !USE_ASYNC, !USE_SHARED_X);
}

const char * itmv_test10() {
  return itmv_test(
      "Test 10: n=4K t=1K 32x128 threads/shared mem",
      !TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<5, 1<<7,
      !USE_ASYNC, USE_SHARED_X);
}
const char * itmv_test10a() {
  return itmv_test(
      "Test 10a: n=4K t=1K 8x128 threads/shared mem",
      !TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<3, 1<<7,
      !USE_ASYNC, USE_SHARED_X);
}

const char * itmv_test11() {
  return itmv_test(
      "Test 11: n=4K t=1K 32x128 threads/Async",
      !TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<5, 1<<7,
      USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test11a() {
  return itmv_test(
      "Test 11a: n=4K t=1K 8x128 threads/Async",
      !TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<3, 1<<7,
      USE_ASYNC, !USE_SHARED_X);
}

const char * itmv_test12() {
  return itmv_test(
      "Test 11: n=4K t=1K 32x128 threads/Async",
      TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<5, 1<<7,
      USE_ASYNC, !USE_SHARED_X);
}
const char * itmv_test12a() {
  return itmv_test(
      "Test 11a: n=4K t=1K 8x128 threads/Async",
      TEST_CORRECTNESS, 4096, !UPPER_TRIANGULAR, 1024, 1<<3, 1<<7,
      USE_ASYNC, !USE_SHARED_X);
}


/*-------------------------------------------------------------------
 * Run all basic tests.  
 */
void run_basic_tests(void){
  printf(">>>>>>>>>>>>>>>>>>>>>>>>>\n");
  printf("Start running itmv tests.\n");
  printf(">>>>>>>>>>>>>>>>>>>>>>>>>\n");

  /*
   * Basic correctness tests without shared memory
   */
  mu_run_test(itmv_test1);
  mu_run_test(itmv_test2);
  mu_run_test(itmv_test3);
  mu_run_test(itmv_test4);


  /*
   * Basic correctness tests with async mode under 1x1 thread config. 
   */
  mu_run_test(itmv_test8a);
  mu_run_test(itmv_test8b);
  mu_run_test(itmv_test8c);
  mu_run_test(itmv_test8d);

}

/*---------------------------------------------------------------------
 * Run tests for larger matrices. 
 * You should call only when your basic tests succeed. 
 */
void run_large_matrix_tests(void){
  /*
   * Large matrix tests without using shared memory
   */
  mu_run_test(itmv_test9);
  mu_run_test(itmv_test9a);
  mu_run_test(itmv_test9b);
  mu_run_test(itmv_test9c); 
  

  /*
   * Large matrix tests with asynchronous mode.
   */
  mu_run_test(itmv_test11);
  mu_run_test(itmv_test11a);
}

void run_large_matrix_tests_with_seq(void) {
  mu_run_test(itmv_test12);
  mu_run_test(itmv_test12a);
}

/*-------------------------------------------------------------------
 * The main entrance to run all tests.
 * Only Proc 0 prints the test summary
 */
int main(){

  run_basic_tests();
  /*You should call large matrix tests only after passing your basic tests.*/
  /*run_large_matrix_tests();*/

  // run_large_matrix_tests_with_seq();

  mu_print_test_summary("\nSummary:");
}
